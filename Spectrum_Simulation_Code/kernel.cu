#include "hip/hip_runtime.h"
#include "globalSwitchStatements.h"
#include "kernel.cuh"

#include "hip/hip_runtime.h"

// #include <stdio.h>
#include <math.h>
#include <vector>
#include <iostream>

#include "vec3d.cuh"
#include "Orbit.cuh"
#include "LISA.h"
#include "dataHandler.h"
#include "hdmem.cuh"

using namespace std;
using namespace cudaUtil;
using namespace lisa;

#ifdef DEBUG_MODE
#define print(input) cout << "kernel: " << input << endl
#else
#ifdef NO_PRINT
#define print(input)
#else
#define print(input) cout << input << endl
#endif
#endif

#define syncThreads __syncthreads()
#define syncDevice hipDeviceSynchronize()

#define THREAD_COUNT 128

#define G 6.67408e-11f

__global__ void coreKernel(Orbit* asteroids,
	vec3d* lisaLocs, vec3d* lisaUnitVectors,
	float time, vec3d* forceRelOutput, vec3d* forceAbsOutput,
	int n, int asteroidCount, int blocksPerSampleTime)
{
	//by sharing memory, a summation can be performed
	//within this method call to save on memory
	__shared__ vec3d tempVals[THREAD_COUNT];
	__shared__ vec3d tempAbsVals[3 * THREAD_COUNT];
	syncThreads;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < (asteroidCount))
	{
		float mu = asteroids[index].getMass() * G;//M*G
		asteroids[index].setTime(time);//update asteroid to current time
		vec3d pos = asteroids[index].pos();//get updated position
		//get vector-wise displacement from each satellite
		vec3d disp1 = lisaLocs[0] - pos;
		vec3d disp2 = lisaLocs[1] - pos;
		vec3d disp3 = lisaLocs[2] - pos;
		//get scalor distance to each satellite
		float dist1 = disp1.magnitude();
		float dist2 = disp2.magnitude();
		float dist3 = disp3.magnitude();
		//get gravitational acceleration on each satellite
		tempAbsVals[3 * threadIdx.x] = -mu * disp1 / (dist1 * dist1 * dist1);
		tempAbsVals[3 * threadIdx.x + 1] = -mu * disp2 / (dist2 * dist2 * dist2);
		tempAbsVals[3 * threadIdx.x + 2] = -mu * disp3 / (dist3 * dist3 * dist3);
		//get projection of accelerations onto each arm
		tempVals[threadIdx.x] =
			vec3d((tempAbsVals[3 * threadIdx.x + 1] - tempAbsVals[3 * threadIdx.x]).dot(lisaUnitVectors[0]),
			(tempAbsVals[3 * threadIdx.x + 2] - tempAbsVals[3 * threadIdx.x]).dot(lisaUnitVectors[1]),
			(tempAbsVals[3 * threadIdx.x + 2] - tempAbsVals[3 * threadIdx.x + 1]).dot(lisaUnitVectors[2]));
	}
	syncThreads;
	int i = blockDim.x;
	//sum over all threads in this block
	while (i != 1)
	{
		int j = i & 1;
		i >>= 1;
		if (threadIdx.x < i)
		{
			tempVals[threadIdx.x] += tempVals[threadIdx.x + i + j];
			tempAbsVals[3 * threadIdx.x] += tempAbsVals[3 * (threadIdx.x + i + j)];
			tempAbsVals[3 * threadIdx.x + 1] += tempAbsVals[3 * (threadIdx.x + i + j) + 1];
			tempAbsVals[3 * threadIdx.x + 2] += tempAbsVals[3 * (threadIdx.x + i + j) + 2];
		}
		i += j;
		syncThreads;
	}
	//set output value
	if (threadIdx.x == 0)
	{
		forceRelOutput[blockIdx.x + (n) * (blocksPerSampleTime)] = tempVals[0];
		forceAbsOutput[3 * (blockIdx.x + (n) * (blocksPerSampleTime))] = tempAbsVals[0];
		forceAbsOutput[3 * (blockIdx.x + (n) * (blocksPerSampleTime)) + 1] = tempAbsVals[1];
		forceAbsOutput[3 * (blockIdx.x + (n) * (blocksPerSampleTime)) + 2] = tempAbsVals[2];
	}
}

__global__ void compactingKernel(
	vec3d* forceRelOutput,
	vec3d* compactForceRelOutput,
	vec3d* forceAbsOutput,
	vec3d* compactForceAbsOutput,
	int sampleTimes, int blocksPerSampleTime)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < sampleTimes)
	{
		compactForceRelOutput[index] = vec3d();
		compactForceAbsOutput[3 * index] = vec3d();
		compactForceAbsOutput[3 * index + 1] = vec3d();
		compactForceAbsOutput[3 * index + 2] = vec3d();
		for (int i = 0; i < (blocksPerSampleTime); i++)
		{
			compactForceRelOutput[index] +=
				forceRelOutput[index * (blocksPerSampleTime) + i];
			compactForceAbsOutput[3 * index] +=
				forceAbsOutput[3 * (index * (blocksPerSampleTime) + i)];
			compactForceAbsOutput[3 * index + 1] +=
				forceAbsOutput[3 * (index * (blocksPerSampleTime) + i) + 1];
			compactForceAbsOutput[3 * index + 2] +=
				forceAbsOutput[3 * (index * (blocksPerSampleTime) + i) + 2];
		}
	}
}

bool executeCudaCode(dataHandler& data)
{
	vector<float> sampleTimes = data.generateSampleTimes();
	vector<Orbit> asteroids = data.generateAsteroids();
	LISA lisa = data.generateLISA();
	//final output vectors
	vector<vec3d> outputRel(sampleTimes.size());
	vector<vector<vec3d>> outputAbs(sampleTimes.size());
	//vector<vector<vec3d>> unitVectors(sampleTimeCount);

	//number of blocks required to simulate a single intance of time
	int blocksPerSampleTime = (asteroids.size() + THREAD_COUNT - 1) / THREAD_COUNT;

	//three vectors for position and three vectors for unit vectors
	//for LISA at each point of time. First dimension specifies time,
	//second dimension specifies which of three.
	vector<vector<vec3d>> lisaLocationsVector(sampleTimes.size());
	vector<vector<vec3d>> lisaUnitVectorsVector(sampleTimes.size());

	for (int i = 0; i < sampleTimes.size(); i++){
		lisa.setTime(sampleTimes[i]);
		lisaLocationsVector[i] = lisa.getPositions();
		lisaUnitVectorsVector[i] = lisa.getUnitVectors(lisaLocationsVector[i]);
	}

	// //device memory instantiated with asteroids vector
	hdmem<Orbit> orbits = hdmem<Orbit>(asteroids).loadMemory().updateDevice();

	//single time slot of lisa data
	hdmem<vec3d> lisaLocations = hdmem<vec3d>(3).loadMemory();
	hdmem<vec3d> lisaUnitVectors = hdmem<vec3d>(3).loadMemory();
	//device memory of output stuff
	hdmem<vec3d> partialRelResults =
		hdmem<vec3d>(sampleTimes.size() * blocksPerSampleTime).loadMemory();
	hdmem<vec3d> partialAbsResults = 
		hdmem<vec3d>(3 * sampleTimes.size() * blocksPerSampleTime).loadMemory();
	hdmem<vec3d> relResults = hdmem<vec3d>(sampleTimes.size()).loadMemory();
	hdmem<vec3d> absResults = hdmem<vec3d>(3 * sampleTimes.size()).loadMemory();

	print((float)(orbits.deviceMemoryAllocated()
		+ lisaLocations.deviceMemoryAllocated()
		+ lisaUnitVectors.deviceMemoryAllocated()
		+ partialRelResults.deviceMemoryAllocated()
		+ partialAbsResults.deviceMemoryAllocated()
		+ relResults.deviceMemoryAllocated()
		+ absResults.deviceMemoryAllocated())/1000000 << " MB Allocated on Device");

	//Core simulation loop
	dim3 grid(blocksPerSampleTime);
	dim3 block(THREAD_COUNT);
	for (int i = 0; i < sampleTimes.size(); i++)
	{
		//update time specific values and check validity of device memory copy
		bool working = true;
		working &= lisaLocations.setAll(lisaLocationsVector[i]).isGood();
		working &= lisaUnitVectors.setAll(lisaUnitVectorsVector[i]).isGood();
		if (!working)
		{
			print("failure on step " << i << ". (0 indexed)");
			print("lisa location status");
			lisaLocations.updateHost();
			print("lisu unit vectors status");
			lisaUnitVectors.updateHost();
			print("orbit status");
			orbits.updateHost();
			print("partial relative results status");
			partialRelResults.updateHost();
			print("partial absolute results status");
			partialAbsResults.updateHost();
			print("relative results status");
			relResults.updateHost();
			print("absolute results status");
			absResults.updateHost();
			// killProgram();
			goto killSpace;
		}
		//simulate single point of time
		coreKernel << <grid, block >> >
			(orbits, lisaLocations, lisaUnitVectors, sampleTimes[i],
				partialRelResults, partialAbsResults,
				i, orbits.size(), blocksPerSampleTime);
		syncDevice;
	}
	//perform final sumation for each point of time.
	grid = dim3((sampleTimes.size() + THREAD_COUNT - 1) / THREAD_COUNT);
	block = dim3(THREAD_COUNT);
	compactingKernel << <grid, block >> >
		(partialRelResults, relResults, partialAbsResults,
			absResults, sampleTimes.size(), blocksPerSampleTime);
	syncDevice;
	//fill in output vectors
	relResults.updateHost();
	absResults.updateHost();
	for (int i = 0; i < sampleTimes.size(); i++)
	{
		outputRel[i] = relResults.get(i);
		for (int j = 0; j < 3; j++)
		{
			outputAbs[i].push_back(absResults.get(3 * i + j));
		}
	}
	data.handOffResults(outputRel, outputAbs, lisaUnitVectorsVector, true);

	return true;
killSpace:
	return false;
}