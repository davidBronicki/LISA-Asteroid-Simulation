#include "hip/hip_runtime.h"
#include "vec3d.cuh"

#include <string>

#include "globalFunctions.h"

#define TOTAL_ACCESS __host__ __device__

using namespace std;
using namespace cudaUtil;

TOTAL_ACCESS vec3d::vec3d()
{
	x = 0;
	y = 0;
	z = 0;
}
TOTAL_ACCESS vec3d::vec3d(float inX, float inY)
{
	x = inX;
	y = inY;
	z = 0;
}
TOTAL_ACCESS vec3d::vec3d(float inX, float inY, float inZ)
{
	x = inX;
	y = inY;
	z = inZ;
}
TOTAL_ACCESS vec3d & vec3d::operator+=(const vec3d rhs)
{
	x += rhs.x;
	y += rhs.y;
	z += rhs.z;
	return *this;
}
TOTAL_ACCESS vec3d & vec3d::operator-=(const vec3d rhs)
{
	x -= rhs.x;
	y -= rhs.y;
	z -= rhs.z;
	return *this;
}
TOTAL_ACCESS vec3d & vec3d::operator*=(const float rhs)
{
	x *= rhs;
	y *= rhs;
	z *= rhs;
	return *this;
}
TOTAL_ACCESS vec3d & vec3d::operator/=(const float & rhs)
{
	x /= rhs;
	y /= rhs;
	z /= rhs;
	return *this;
}
string vec3d::toString()
{
	return "< " + str(x) + "," + str(y)
		+ "," + str(z) + " >";
}
TOTAL_ACCESS float vec3d::dot(vec3d otherVector)
{
	return x * otherVector.x + y * otherVector.y + z * otherVector.z;
}
TOTAL_ACCESS float vec3d::magnitude()
{
	return sqrt(x*x + y*y + z*z);
}
TOTAL_ACCESS float vec3d::squared()
{
	return x*x + y*y + z*z;
}
TOTAL_ACCESS vec3d vec3d::normalized()
{
	return vec3d(*this) / magnitude();
}
vector<float> vec3d::toFloatVector()
{
	vector<float> output(3);
	output[0] = x;
	output[1] = y;
	output[2] = z;
	return output;
}

TOTAL_ACCESS vec3d cudaUtil::operator+(vec3d lhs, const vec3d & rhs)
{
	lhs += rhs;
	return lhs;
}
TOTAL_ACCESS vec3d cudaUtil::operator-(vec3d lhs, const vec3d & rhs)
{
	lhs -= rhs;
	return lhs;
}
TOTAL_ACCESS vec3d cudaUtil::operator*(vec3d lhs, const float & rhs)
{
	lhs *= rhs;
	return lhs;
}
TOTAL_ACCESS vec3d cudaUtil::operator*(const float & lhs, vec3d rhs)
{
	rhs *= lhs;
	return rhs;
}
TOTAL_ACCESS vec3d cudaUtil::operator/(vec3d lhs, const float & rhs)
{
	lhs /= rhs;
	return lhs;
}
