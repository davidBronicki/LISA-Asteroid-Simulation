#include "hip/hip_runtime.h"
#include "Orbit.cuh"

#include <math.h>
#include <iostream>

#include "vec3d.cuh"

using namespace std;
using namespace lisa;
using namespace cudaUtil;

#define makeR r = semiLatusRectum / (1 + eccentricity * cos(trueAnomaly))
#define totalAngle (trueAnomaly + argumentOfPeriapsis)
#define eccToMeanAnom meanAnomaly = eccentricAnomaly - eccentricity * sin(eccentricAnomaly)
#define difOfMean(psi) (meanAnomaly - psi + eccentricity * sin(psi))

#define print(input) cout << input << endl

#define TOLERANCE 1e-10f
#define PI 3.1415926535898f
#define TWOPI 6.2831853071796f
#define au 149597870700.0f
#define G 6.67408e-11f
#define earthMass 5.972e24f
#define solarMass 1.989e30f
#define mu 1.32747e20f
#define TOTAL_ACCESS __host__ __device__

TOTAL_ACCESS float floatMod(float value, float mod)
{
	float extra = value >= 0 ?
		mod * (int)(value / mod) :
		mod * (int)(value / mod - 1);
	return value - extra;
}

Orbit::Orbit()
{
	trueAnomaly = 0;
	eccentricAnomaly = 0;
	meanAnomaly = 0;
	meanAnomaly0 = 0;
	semiMajorAxis = au;
	semiLatusRectum = au;
	eccentricity = 0;
	inclination = 0;
	longitudeOfAscendingNode = 0;
	argumentOfPeriapsis = 0;
	meanAngularMotion = 1.991e-7f;
	cLongAsc = 1;
	sLongAsc = 0;
	cInc = 1;
	sInc = 1e-7f;
	r = au;

	time = 0;
	time0 = 0;
	mass = 1;
	name = "default name";
}

Orbit::Orbit(float anomaly, float ecc, float semiMajor,
	float incline, float longAscend,
	float argPeri, bool isTrueAnom)
{
	eccentricity = ecc;
	if (isTrueAnom)
	{
		trueAnomaly = floatMod(anomaly, TWOPI);
		TrueToEccAnom();
		eccToMeanAnom;
	}
	else
	{
		meanAnomaly = floatMod(anomaly, TWOPI);
		MeanToEccAnom();
		EccToTrueAnom();
	}
	meanAnomaly0 = meanAnomaly;
	semiMajorAxis = semiMajor;
	semiLatusRectum = semiMajor * (1 - ecc * ecc);
	inclination = incline;
	longitudeOfAscendingNode = floatMod(longAscend, TWOPI);
	argumentOfPeriapsis = floatMod(argPeri, TWOPI);
	meanAngularMotion = sqrt(mu / (semiMajorAxis * semiMajorAxis * semiMajorAxis));
	cLongAsc = cos(longitudeOfAscendingNode);
	sLongAsc = sin(longitudeOfAscendingNode);
	cInc = cos(inclination);
	sInc = sin(inclination);
	if (sInc == 0)
	{
		sInc = 1e-7;
	}
	makeR;

	mass = 1;
	name = "default name";
}
Orbit::Orbit(float anomaly, float ecc, float semiMajor,
	float incline, float longAscend,
	float argPeri, bool isTrueAnom, float timeOfParameters,
	float definedTimeZero, float inMass, std::string inName)
{
	eccentricity = ecc;
	if (isTrueAnom)
	{
		trueAnomaly = floatMod(anomaly, TWOPI);
		TrueToEccAnom();
		eccToMeanAnom;
	}
	else
	{
		meanAnomaly = floatMod(anomaly, TWOPI);
		MeanToEccAnom();
		EccToTrueAnom();
	}
	semiMajorAxis = semiMajor;
	semiLatusRectum = semiMajor * (1 - ecc * ecc);
	inclination = incline;
	longitudeOfAscendingNode = floatMod(longAscend, TWOPI);
	argumentOfPeriapsis = floatMod(argPeri, TWOPI);
	meanAngularMotion = sqrt(mu / (semiMajorAxis * semiMajorAxis * semiMajorAxis));
	cLongAsc = cos(longitudeOfAscendingNode);
	sLongAsc = sin(longitudeOfAscendingNode);
	cInc = cos(inclination);
	sInc = sin(inclination);
	if (sInc == 0)
	{
		sInc = 1e-7;
	}
	makeR;

	meanAnomaly0 = meanAnomaly + (definedTimeZero - timeOfParameters) * meanAngularMotion;
	meanAnomaly = meanAnomaly0;
	MeanToEccAnom();
	EccToTrueAnom();
	time0 = definedTimeZero;
	time = definedTimeZero;
	mass = inMass;
	name = inName;
}

TOTAL_ACCESS void Orbit::TrueToEccAnom()
{
	float temp = sqrt((1 - eccentricity) / (1 + eccentricity)) * tan(trueAnomaly / 2);
	temp = 2 * atan(temp);
	eccentricAnomaly = temp >= 0 ? temp : temp + TWOPI;
	//eccentricAnomaly = 2 * atan(temp);
}
TOTAL_ACCESS void Orbit::EccToTrueAnom()
{
	float temp = sqrt((1 + eccentricity) / (1 - eccentricity)) * tan(eccentricAnomaly / 2);
	temp = 2 * atan(temp);
	trueAnomaly = temp > 0 ? temp : temp + TWOPI;
	//trueAnomaly = 2 * atan(temp);
}
TOTAL_ACCESS void Orbit::MeanToEccAnom()
{
	//int temp = (int)(meanAnomaly / TWOPI);
	//meanAnomaly -= temp * TWOPI;
	double lower = 0;
	double upper = TWOPI;
	double width = TWOPI;
	double midPoint = PI;
	while (width > TOLERANCE)
	{
		if (difOfMean(midPoint) * difOfMean(lower) > 0)
		{
			lower = midPoint;
		}
		else
		{
			upper = midPoint;
		}
		width = upper - lower;
		midPoint = (upper + lower) / 2;
	}
	eccentricAnomaly = midPoint;
}

TOTAL_ACCESS void Orbit::setTime(float newTime)
{
	time = newTime;
	//meanAnomaly = meanAnomaly0 + (time - time0) * meanAngularMotion;
	meanAnomaly = floatMod(meanAnomaly0 + (time * meanAngularMotion), TWOPI);
	MeanToEccAnom();
	EccToTrueAnom();
	makeR;
}

TOTAL_ACCESS void Orbit::setTime(float newTime, bool useAbsoluteTime){
	if (useAbsoluteTime){
		time = newTime;
		//meanAnomaly = meanAnomaly0 + (time - time0) * meanAngularMotion;
		meanAnomaly = floatMod(meanAnomaly0 + ((time - time0) * meanAngularMotion), TWOPI);
		MeanToEccAnom();
		EccToTrueAnom();
		makeR;
	}
	else setTime(newTime);
}

TOTAL_ACCESS float Orbit::x()
{
	return r * (cLongAsc * cos(totalAngle) - cInc * sLongAsc * sin(totalAngle));
}

TOTAL_ACCESS float Orbit::y()
{
	return r * (sLongAsc * cos(totalAngle) + cInc * cLongAsc * sin(totalAngle));
}

TOTAL_ACCESS float Orbit::z()
{
	return r * (sInc * sin(totalAngle));
}

TOTAL_ACCESS float Orbit::getMass()
{
	return mass;
}

TOTAL_ACCESS vec3d Orbit::pos()
{
	return vec3d(x(), y(), z());
}

string Orbit::getName()
{
	return name;
}

